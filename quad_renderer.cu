#include "hip/hip_runtime.h"
#include "quad_renderer.h"

#define TILE_SIZE 3

#ifdef __HIPCC__
#define HOST __host__
#define DEVICE __device__
#define GLOBAL __global__
#else
#define HOST
#define DEVICE
#define GLOBAL
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#endif

namespace cuda_renderer
{

HOST DEVICE unsigned DivideCeil(unsigned dividend, unsigned divisor)
{
	return 1 + ((dividend - 1) / divisor);
}

DEVICE Vec VecCreate(float x, float y, float z);
DEVICE Vec VecAdd(Vec a, Vec b);
DEVICE Vec VecSub(Vec A, Vec B);
DEVICE float DotProduct(Vec A, Vec B);
DEVICE float VecLen(Vec a);
DEVICE Vec VecMul(Vec v,float t);
DEVICE Vec VecUnit(Vec a);
DEVICE Vec lerp(Vec v0, Vec v1, float t);
DEVICE PointProjection Proj(Vec t, Camera k, bool force);
DEVICE extern float rainbow[6][3];

DEVICE static float calculateSignedArea(const float2 tri[3])
{
	return 0.5 * ((tri[2].x - tri[0].x) * (tri[1].y - tri[0].y) - (tri[1].x - tri[0].x) * (tri[2].y - tri[0].y));
}

DEVICE static float calculateBarycentricValue(float2 a, float2 b, float2 c, const float2 tri[3])
{
	float2 baryTri[3] = { a, b, c };
	return calculateSignedArea(baryTri) / calculateSignedArea(tri);
}

DEVICE static float3 calculateBarycentric(const float2 tri[3], float2 point)
{
	float beta = calculateBarycentricValue(make_float2(tri[0].x, tri[0].y), point, make_float2(tri[2].x, tri[2].y), tri);
	float gamma = calculateBarycentricValue(make_float2(tri[0].x, tri[0].y), make_float2(tri[1].x, tri[1].y), point, tri);
	float alpha = 1.0 - beta - gamma;
	return make_float3(alpha, beta, gamma);
}

DEVICE static bool isBarycentricCoordInBounds(const float3 barycentricCoord)
{
	return barycentricCoord.x >= 0.0 && barycentricCoord.x <= 1.0 &&
				 barycentricCoord.y >= 0.0 && barycentricCoord.y <= 1.0 &&
				 barycentricCoord.z >= 0.0 && barycentricCoord.z <= 1.0;
}

template <typename T> DEVICE static T combineBarycentric(float3 bary, T values[3])
{
	return values[0] * bary.x +
				 values[1] * bary.y +
				 values[2] * bary.z;
}

template <> DEVICE static float2 combineBarycentric(float3 bary, float2 values[3])
{
	return make_float2(
					values[0].x * bary.x +
					values[1].x * bary.y +
					values[2].x * bary.z,
					values[0].y * bary.x +
					values[1].y * bary.y +
					values[2].y * bary.z);
}

GLOBAL
void QuadRenderer(
	Camera cam,
	float *quadData,
	SQuadBufDesc *quadBufDesc,
	int numSlices,
	int numTiles,
	unsigned *intensityRaster,
	int screenW,
	int screenH,
	float brightnessMultiplier,
	float maxLength,
	float scale
	)
{
	unsigned blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
	unsigned i = __mul24(blockId, blockDim.x) + threadIdx.x;

	const unsigned iSlice = i / numTiles;
	if (iSlice >= numSlices) return;

	SQuadBufDesc *pDesc = quadBufDesc + iSlice;

	PointProjection vertices[4] =
	{
		Proj(VecMul(pDesc->vertices[0], scale), cam, true),
		Proj(VecMul(pDesc->vertices[1], scale), cam, true),
		Proj(VecMul(pDesc->vertices[2], scale), cam, true),
		Proj(VecMul(pDesc->vertices[3], scale), cam, true),
	};

	if (vertices[0].zdistRec <= 0 && vertices[1].zdistRec <= 0 && vertices[2].zdistRec <= 0 && vertices[3].zdistRec <= 0) return;

	unsigned iTile = i % numTiles;

	struct
	{
		float2 min;
		float2 max;
	} boundingBox =
	{
		{
			fminf(fminf(vertices[0].x, vertices[1].x), fminf(vertices[2].x, vertices[3].x)),
			fminf(fminf(vertices[0].y, vertices[1].y), fminf(vertices[2].y, vertices[3].y)),
		},
		{
			fmaxf(fmaxf(vertices[0].x, vertices[1].x), fmaxf(vertices[2].x, vertices[3].x)),
			fmaxf(fmaxf(vertices[0].y, vertices[1].y), fmaxf(vertices[2].y, vertices[3].y)),
		},
	};

	unsigned bbWidth = min((unsigned)ceil(boundingBox.max.x), screenW) - max((unsigned)floor(boundingBox.min.x), 0);
	unsigned bbHeight = min((unsigned)ceil(boundingBox.max.y), screenH) - max((unsigned)floor(boundingBox.min.y), 0);
	unsigned tileX = iTile % DivideCeil(bbWidth, TILE_SIZE);
	unsigned tileY = iTile / DivideCeil(bbWidth, TILE_SIZE);
	if (tileY >= DivideCeil(bbHeight, TILE_SIZE)) return;

	int minX = max((int)round(boundingBox.min.x), 0) + tileX * TILE_SIZE;
	int minY = max((int)round(boundingBox.min.y), 0) + tileY * TILE_SIZE;
	int maxX = min(minX + TILE_SIZE, screenW);
	int maxY = min(minY + TILE_SIZE, screenH);

	float2 texCoords[] =
	{
		make_float2(0, pDesc->gridWidth * vertices[0].zdistRec),
		make_float2(0, 0),
		make_float2(pDesc->gridWidth * vertices[2].zdistRec, pDesc->gridHeight * vertices[2].zdistRec),
		make_float2(pDesc->gridWidth * vertices[3].zdistRec, 0),
	};

	for (int y = minY; y < maxY; ++y)
	{
		for (int x = minX; x < maxX; ++x)
		{
			float2 tri[] =
			{
				make_float2(vertices[0].x, vertices[0].y),
				make_float2(vertices[1].x, vertices[1].y),
				make_float2(vertices[2].x, vertices[2].y),
				make_float2(vertices[3].x, vertices[3].y),
			};
			float3 bary1 = calculateBarycentric(tri, make_float2(x, y));
			float3 bary2 = calculateBarycentric(tri + 1, make_float2(x, y));
			PointProjection *pVerts = nullptr;
			float2 *pTexCoords;
			if (isBarycentricCoordInBounds(bary1))
			{
				pVerts = vertices;
				pTexCoords = texCoords;
			}
			else if (isBarycentricCoordInBounds(bary2))
			{
				pVerts = vertices + 1;
				pTexCoords = texCoords + 1;
				bary1 = bary2;
			}

			if (pVerts)
			{
				float zs[] = { pVerts[0].zdistRec, pVerts[1].zdistRec, pVerts[2].zdistRec };
				float z = combineBarycentric(bary1, zs);
				if (z > 0)
				{
					int dstIndex = (x + y * screenW) * 3;
					float2 texCoord = combineBarycentric(bary1, pTexCoords);
					texCoord.x /= z;
					texCoord.y /= z;
					if (texCoord.x >= 0 && texCoord.y >= 0)
					{
						int texIndex = min((int)texCoord.y, pDesc->gridHeight) * pDesc->gridWidth + min((int)texCoord.x, pDesc->gridWidth);
						float length = quadData[pDesc->dataOffset + texIndex];
						float len = min(length / maxLength * 4, 4.f);
						int rainbowIndex = (int)len;
						float fade = len - rainbowIndex;
						Vec colorFrom = VecCreate(rainbow[rainbowIndex][0], rainbow[rainbowIndex][1], rainbow[rainbowIndex][2]);
						Vec colorTo = VecCreate(rainbow[rainbowIndex + 1][0], rainbow[rainbowIndex + 1][1], rainbow[rainbowIndex + 1][2]);
						Vec color = lerp(colorFrom, colorTo, fade);
						float brightness = length * length * 10 * z * z * brightnessMultiplier;

						intensityRaster[dstIndex]     += unsigned(brightness * color.x);
						intensityRaster[dstIndex + 1] += unsigned(brightness * color.y);
						intensityRaster[dstIndex + 2] += unsigned(brightness * color.z);
					}
				}
			}
		}
	}
}

}

void CallQuadRenderer(
	Camera cam,
	float *quadData,
	SQuadBufDesc *quadBufDesc,
	int numSlices,
	unsigned *intensityRaster,
	int screenW,
	int screenH,
	float brightnessMultiplier,
	float maxLength,
	float scale
	)
{
	if (numSlices < 1) return;

	unsigned numTiles = cuda_renderer::DivideCeil(screenW, TILE_SIZE) * cuda_renderer::DivideCeil(screenH, TILE_SIZE);
	unsigned numThreads = numTiles * numSlices;
	unsigned threadGroupSize = 128;
	dim3 threadGroups(cuda_renderer::DivideCeil(numThreads, threadGroupSize));
	while (threadGroups.x > 65535)
	{
		threadGroups.x /= 2;
		threadGroups.y *= 2;
	}

	cuda_renderer::QuadRenderer<<<threadGroups, threadGroupSize>>>(
		cam,
		quadData,
		quadBufDesc,
		numSlices,
		numTiles,
		intensityRaster,
		screenW,
		screenH,
		brightnessMultiplier,
		maxLength,
		scale
		);
}

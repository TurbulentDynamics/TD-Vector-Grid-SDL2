#include "hip/hip_runtime.h"
#include "renderer.h"

#define TILE_SIZE 3

#ifdef __HIPCC__
#define HOST __host__
#define DEVICE __device__
#define GLOBAL __global__
#else
#define HOST
#define DEVICE
#define GLOBAL
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#endif

const unsigned threadN=65536/4;

namespace cuda_renderer
{
DEVICE
Vec VecCreate(float x, float y, float z)
{
	Vec res;
	res.x=x;
	res.y=y;
	res.z=z;
	return res;
}

DEVICE
Vec VecAdd(Vec a, Vec b)
{
	return VecCreate(a.x + b.x, a.y + b.y, a.z + b.z);
}

DEVICE
Vec VecSub(Vec A, Vec B)
{
	A.x-=B.x;
	A.y-=B.y;
	A.z-=B.z;
	return A;
}

DEVICE
float DotProduct(Vec A, Vec B)
{
	return A.x*B.x + A.y*B.y + A.z*B.z;
}

DEVICE
float VecLen(Vec a)
{
	return sqrt(DotProduct(a,a));
}
DEVICE
Vec VecMul(Vec v,float t)
{
	v.x*=t;
	v.y*=t;
	v.z*=t;
	return v;
}
DEVICE
Vec VecUnit(Vec a)
{
	return VecMul(a, 1.0f/VecLen(a));
}

DEVICE
Vec lerp(Vec v0, Vec v1, float t)
{
		return VecAdd(VecMul(v0, (1-t)), VecMul(v1, t));
}

DEVICE static float calculateSignedArea(const float2 tri[3])
{
	return 0.5 * ((tri[2].x - tri[0].x) * (tri[1].y - tri[0].y) - (tri[1].x - tri[0].x) * (tri[2].y - tri[0].y));
}

DEVICE static float calculateBarycentricValue(float2 a, float2 b, float2 c, const float2 tri[3])
{
	float2 baryTri[3] = { a, b, c };
	return calculateSignedArea(baryTri) / calculateSignedArea(tri);
}

DEVICE static float3 calculateBarycentric(const float2 tri[3], float2 point)
{
	float beta = calculateBarycentricValue(make_float2(tri[0].x, tri[0].y), point, make_float2(tri[2].x, tri[2].y), tri);
	float gamma = calculateBarycentricValue(make_float2(tri[0].x, tri[0].y), make_float2(tri[1].x, tri[1].y), point, tri);
	float alpha = 1.0 - beta - gamma;
	return make_float3(alpha, beta, gamma);
}

DEVICE static bool isBarycentricCoordInBounds(const float3 barycentricCoord)
{
	return barycentricCoord.x >= 0.0 && barycentricCoord.x <= 1.0 &&
				 barycentricCoord.y >= 0.0 && barycentricCoord.y <= 1.0 &&
				 barycentricCoord.z >= 0.0 && barycentricCoord.z <= 1.0;
}

template <typename T> DEVICE static T combineBarycentric(float3 bary, T values[3])
{
	return values[0] * bary.x +
				 values[1] * bary.y +
				 values[2] * bary.z;
}

template <> DEVICE float2 combineBarycentric(float3 bary, float2 values[3])
{
	return make_float2(
					values[0].x * bary.x +
					values[1].x * bary.y +
					values[2].x * bary.z,
					values[0].y * bary.x +
					values[1].y * bary.y +
					values[2].y * bary.z);
}


HOST DEVICE unsigned DivideCeil(unsigned dividend, unsigned divisor)
{
	return 1 + ((dividend - 1) / divisor);
}

DEVICE float rainbow[][3] =
{
	{ 1.0f, 0.0f, 0.0f },
	{ 1.0f, 1.0f, 0.0f },
	{ 0.0f, 1.0f, 0.0f },
	{ 0.0f, 1.0f, 1.0f },
	{ 0.0f, 0.0f, 1.0f },
	{ 1.0f, 0.0f, 1.0f },
};

DEVICE bool useOrtho = false;

DEVICE
PointProjection Proj(Vec t, Camera k, bool force = false)
{
	PointProjection ret;
	Vec diff=VecSub(t,k.eye);
	float zdist = DotProduct(diff, k.dir);

	if (!force && zdist < 0.1f) {
		ret.zdistRec = -1;
		return ret;
		}
	ret.zdistRec=1.0f / (useOrtho ? VecLen(k.eye) : zdist);
	Vec proj=VecMul(diff, k.screenDist * ret.zdistRec);
	proj =VecAdd(proj, k.upLeftCornerTrans);
	ret.x = DotProduct(proj, k.xd);
	ret.y = DotProduct(proj, k.yd);
	return ret;
}

GLOBAL
void MovingPointsRenderer(
		Camera cam,
		float* mpData,
		int mpN,
		SMpBufDesc* bufDesc,
		int bufN,
		unsigned* intensityRaster,
		int screenW, int screenH,
		unsigned curtime,
		float brightnessMultiplier,
		float lengthMultiplier,
		float maxLength,
		bool useColor,
		bool useSpeed
		)
{
	const unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;

	int bufI=0;
	for (unsigned i=idx; true; i+=threadN)
		{
		while (i>=bufDesc[bufI].n) {
			i -= bufDesc[bufI].n;
			bufI++;
			if (bufI>=bufN)
				return;
			}
		int pitch = bufDesc[bufI].n;
		const unsigned timeMs = curtime - bufDesc[bufI].startTime;
		float* data =&mpData[bufDesc[bufI].beg + i];

		float mpBegX = data[pitch*0];
		float mpBegY = data[pitch*1];
		float mpBegZ = data[pitch*2];
		float mpVelX = data[pitch*3];
		float mpVelY = data[pitch*4];
		float mpVelZ = data[pitch*5];
		float mpOffs = data[pitch*6];
		float mpBrig = data[pitch*7];

		Vec beg=VecCreate(mpBegX,mpBegY,mpBegZ);
		Vec v  =VecCreate(mpVelX,mpVelY,mpVelZ);
		//Vec v(1,1,1);
		float len = VecLen(v) ;
		float pos =mpOffs + timeMs * (useSpeed ? (len/500.0f) : (1.0/4000));
		if (pos>1) continue;
		Vec p = VecAdd(beg,VecMul(v,pos*lengthMultiplier));

		PointProjection proj = Proj(p,cam);

		if (proj.zdistRec<=0) continue;
		float brightness = mpBrig * proj.zdistRec * proj.zdistRec * brightnessMultiplier;

		int x = int(proj.x);
		int y = int(proj.y);

		if (x<0 || x>=screenW) continue;
		if (y<0 || y>=screenH) continue;
		int dstIndex = (x + y*screenW) * 3;
		len = min(len/ maxLength * 4, 4.f);
		int rainbowIndex = (int)len;
		float fade = len - rainbowIndex;
		Vec colorFrom = Vec(rainbow[rainbowIndex][0], rainbow[rainbowIndex][1], rainbow[rainbowIndex][2]);
		Vec colorTo = Vec(rainbow[rainbowIndex + 1][0], rainbow[rainbowIndex + 1][1], rainbow[rainbowIndex + 1][2]);
		Vec color = useColor ? lerp(colorFrom, colorTo, fade) : Vec(1, 1, 1);

		intensityRaster[dstIndex]     += unsigned(brightness * color.x);
		intensityRaster[dstIndex + 1] += unsigned(brightness * color.y);
		intensityRaster[dstIndex + 2] += unsigned(brightness * color.z);

		//intensityRaster[i%(100*1000)] += i;
		}
}

GLOBAL
void QuadRenderer(
	Camera cam,
	float *quadData,
	SQuadBufDesc *quadBufDesc,
	int numSlices,
	int numTiles,
	unsigned *intensityRaster,
	int screenW,
	int screenH,
	float brightnessMultiplier,
	float maxLength,
	float scale
	)
{
	unsigned blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
	unsigned i = __mul24(blockId, blockDim.x) + threadIdx.x;

	const unsigned iSlice = i / numTiles;
	if (iSlice >= numSlices) return;

	SQuadBufDesc *pDesc = quadBufDesc + iSlice;

	PointProjection vertices[4] =
	{
		Proj(VecMul(pDesc->vertices[0], scale), cam, true),
		Proj(VecMul(pDesc->vertices[1], scale), cam, true),
		Proj(VecMul(pDesc->vertices[2], scale), cam, true),
		Proj(VecMul(pDesc->vertices[3], scale), cam, true),
	};

	if (vertices[0].zdistRec <= 0 && vertices[1].zdistRec <= 0 && vertices[2].zdistRec <= 0 && vertices[3].zdistRec <= 0) return;

	unsigned iTile = i % numTiles;

	struct
	{
		float2 min;
		float2 max;
	} boundingBox =
	{
		{
			fminf(fminf(vertices[0].x, vertices[1].x), fminf(vertices[2].x, vertices[3].x)),
			fminf(fminf(vertices[0].y, vertices[1].y), fminf(vertices[2].y, vertices[3].y)),
		},
		{
			fmaxf(fmaxf(vertices[0].x, vertices[1].x), fmaxf(vertices[2].x, vertices[3].x)),
			fmaxf(fmaxf(vertices[0].y, vertices[1].y), fmaxf(vertices[2].y, vertices[3].y)),
		},
	};

	unsigned bbWidth = min((unsigned)ceil(boundingBox.max.x), screenW) - max((unsigned)floor(boundingBox.min.x), 0);
	unsigned bbHeight = min((unsigned)ceil(boundingBox.max.y), screenH) - max((unsigned)floor(boundingBox.min.y), 0);
	unsigned tileX = iTile % DivideCeil(bbWidth, TILE_SIZE);
	unsigned tileY = iTile / DivideCeil(bbWidth, TILE_SIZE);
	if (tileY >= DivideCeil(bbHeight, TILE_SIZE)) return;

	int minX = max((int)round(boundingBox.min.x), 0) + tileX * TILE_SIZE;
	int minY = max((int)round(boundingBox.min.y), 0) + tileY * TILE_SIZE;
	int maxX = min(minX + TILE_SIZE, screenW);
	int maxY = min(minY + TILE_SIZE, screenH);

	float2 texCoords[] =
	{
		make_float2(0, pDesc->gridWidth * vertices[0].zdistRec),
		make_float2(0, 0),
		make_float2(pDesc->gridWidth * vertices[2].zdistRec, pDesc->gridHeight * vertices[2].zdistRec),
		make_float2(pDesc->gridWidth * vertices[3].zdistRec, 0),
	};

	for (int y = minY; y < maxY; ++y)
	{
		for (int x = minX; x < maxX; ++x)
		{
			float2 tri[] =
			{
				make_float2(vertices[0].x, vertices[0].y),
				make_float2(vertices[1].x, vertices[1].y),
				make_float2(vertices[2].x, vertices[2].y),
				make_float2(vertices[3].x, vertices[3].y),
			};
			float3 bary1 = calculateBarycentric(tri, make_float2(x, y));
			float3 bary2 = calculateBarycentric(tri + 1, make_float2(x, y));
			PointProjection *pVerts = nullptr;
			float2 *pTexCoords;
			if (isBarycentricCoordInBounds(bary1))
			{
				pVerts = vertices;
				pTexCoords = texCoords;
			}
			else if (isBarycentricCoordInBounds(bary2))
			{
				pVerts = vertices + 1;
				pTexCoords = texCoords + 1;
				bary1 = bary2;
			}

			if (pVerts)
			{
				float zs[] = { pVerts[0].zdistRec, pVerts[1].zdistRec, pVerts[2].zdistRec };
				float z = combineBarycentric(bary1, zs);
				if (z > 0)
				{
					int dstIndex = (x + y * screenW) * 3;
					float2 texCoord = combineBarycentric(bary1, pTexCoords);
					texCoord.x /= z;
					texCoord.y /= z;
					if (texCoord.x >= 0 && texCoord.y >= 0)
					{
						int texIndex = min((int)texCoord.y, pDesc->gridHeight) * pDesc->gridWidth + min((int)texCoord.x, pDesc->gridWidth);
						float length = quadData[pDesc->dataOffset + texIndex];
						float len = min(length / maxLength * 4, 4.f);
						int rainbowIndex = (int)len;
						float fade = len - rainbowIndex;
						Vec colorFrom = VecCreate(rainbow[rainbowIndex][0], rainbow[rainbowIndex][1], rainbow[rainbowIndex][2]);
						Vec colorTo = VecCreate(rainbow[rainbowIndex + 1][0], rainbow[rainbowIndex + 1][1], rainbow[rainbowIndex + 1][2]);
						Vec color = lerp(colorFrom, colorTo, fade);
						float brightness = length * length * 10 * z * z * brightnessMultiplier;

						intensityRaster[dstIndex]     += unsigned(brightness * color.x);
						intensityRaster[dstIndex + 1] += unsigned(brightness * color.y);
						intensityRaster[dstIndex + 2] += unsigned(brightness * color.z);
					}
				}
			}
		}
	}
}

} //namespace cuda_renderer ends

void SetOrtho(bool ortho)
{
	hipMemcpyToSymbol(HIP_SYMBOL(cuda_renderer::useOrtho), &ortho, sizeof(ortho));
}

void CallMovingPointsRenderer(
		Camera cam,
		float* mpData,
		SMpBufDesc* bufDesc,
		int bufN,
		unsigned* intensityRaster,
		int screenW, int screenH,
		unsigned curtime,
		float brightnessMultiplier,
		float lengthMultiplier,
		float maxLength,
		bool useColor,
		bool useSpeed
		)
{
	dim3 block(64);
	dim3 grid((unsigned int)ceil(threadN/(float)block.x));

	if (bufN<1) return;

	cuda_renderer::MovingPointsRenderer<<<grid, block>>>(
		cam,
		mpData,
		0,
		bufDesc,
		bufN,
		intensityRaster,
		screenW, screenH,
		curtime,
		brightnessMultiplier,
		lengthMultiplier,
		maxLength,
		useColor,
		useSpeed
		);
}


void CallQuadRenderer(
	Camera cam,
	float *quadData,
	SQuadBufDesc *quadBufDesc,
	int numSlices,
	unsigned *intensityRaster,
	int screenW,
	int screenH,
	float brightnessMultiplier,
	float maxLength,
	float scale
	)
{
	if (numSlices < 1) return;

	unsigned numTiles = cuda_renderer::DivideCeil(screenW, TILE_SIZE) * cuda_renderer::DivideCeil(screenH, TILE_SIZE);
	unsigned numThreads = numTiles * numSlices;
	unsigned threadGroupSize = 128;
	dim3 threadGroups(cuda_renderer::DivideCeil(numThreads, threadGroupSize));
	while (threadGroups.x > 65535)
	{
		threadGroups.x /= 2;
		threadGroups.y *= 2;
	}

	cuda_renderer::QuadRenderer<<<threadGroups, threadGroupSize>>>(
		cam,
		quadData,
		quadBufDesc,
		numSlices,
		numTiles,
		intensityRaster,
		screenW,
		screenH,
		brightnessMultiplier,
		maxLength,
		scale
		);
}

